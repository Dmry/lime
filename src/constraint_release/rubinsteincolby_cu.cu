#include "hip/hip_runtime.h"
#include <vector>
#include <memory>
#include <iostream>
#include <stdexcept>

#include "rubinsteincolby_cu.hpp"

__global__ void
me(float epsilon, int realization_size, float* km, unsigned int* res, unsigned int max)
{
    unsigned int blk = blockIdx.x;
    unsigned int idx = threadIdx.x;
    const size_t stride = (blk*blockDim.x + idx)* realization_size;

    __shared__ unsigned int sres[1];

    if (idx < blockDim.x)
        sres[blk] = 0;
    if (idx == blockDim.x+1)
        res[0] = 0;

    __syncthreads();

    if (stride+realization_size < max)
    {
        float s{0.0};

        s = km[0+stride] + km[1+stride] - epsilon;
        if (s < 0.0) atomicAdd(&sres[blk], 1);

        for (size_t i = 1+stride; i < stride+realization_size-1 ; ++i)
        {
            s = km[i] + km[i+1] - epsilon - (km[i]*km[i])/s;
            if (s < 0.0) atomicAdd(&sres[blk], 1);
        }
    }
    __syncthreads();

    if (stride < blockDim.x)
        atomicAdd(&res[0], sres[blk]);

}

Cu_me_details::Cu_me_details()
: size_{0}, km_ptr_{nullptr}
{
    hipMalloc((void**)&res_, sizeof(unsigned int));
    hipHostMalloc((void**)&h_res_, sizeof(unsigned int));
}

Cu_me_details::~Cu_me_details()
{
    hipFree(km_ptr_);
    hipFree(res_);
    hipHostFree(h_res_);
}

Cu_me_details::Cu_me_details(const Cu_me_details& other)
: size_{other.size_}
{
    hipMalloc((void**)&km_ptr_, static_cast<int>(size_));
    if ( hipSuccess != hipGetLastError() )
        std::cout << "asdf" << std::endl;
    hipMemcpy(km_ptr_, other.km_ptr_, size_, hipMemcpyDeviceToDevice);
}

void
Cu_me_details::set_km(const std::vector<double>& vec)
{
    const std::vector<float> km_vec(vec.begin(), vec.end());

    hipFree(km_ptr_);

    size_ = km_vec.size()*sizeof(float);

    hipMalloc((void**)&km_ptr_, size_);

    hipMemcpy(km_ptr_, km_vec.data(), size_, hipMemcpyHostToDevice);
}

double
Cu_me_details::cu_me(double epsilon, size_t realization_size, size_t realizations) const
{
    me<<<1, realizations>>>(static_cast<float>(epsilon), static_cast<int>(realization_size), km_ptr_, res_, static_cast<unsigned int>(realization_size*realizations));

    hipMemcpy(h_res_, res_, sizeof(unsigned int), hipMemcpyDeviceToHost);

    return static_cast<double>(*h_res_)/static_cast<double>(realization_size*realizations);
}